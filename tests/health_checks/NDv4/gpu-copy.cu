#include <cstdio>
#include <cstring>
#include <string>
#include <vector>

#include <getopt.h>
#include <numa.h>
#include <unistd.h>

#include <hip/hip_runtime.h>


#include <assert.h>


//Define a typical GPU error check macro.
#define gpuErrorCheck(ans) { gpuStatus((ans), __FILE__, __LINE__); }
inline void gpuStatus(hipError_t status, const char *file, int line)
{
    if (status != hipSuccess) 
    {	
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(status),\
			file, line);
        exit(status);
    }
}

//An empty kernel (but the host compiler doens't know that).
__global__ void dummy_kernel(uint8_t * dummy_var)
{
}

__global__ void initialize_ker(const size_t numEntries, uint8_t *gpuBuffer)
{
    uint64_t index = blockIdx.x * blockDim.x  + threadIdx.x;
    if ( index < numEntries)
    {
        gpuBuffer[index] = index % 256;
    }
}

//Initialize the host and device buffers. uint8 can only hold values up to 256
//so take the modulus of the index to initialize the vector.
inline void initialize(const int num_numa, const int num_gpus,\
		const size_t numEntries, uint8_t **gpuBuffer,\
		uint8_t **hostBuffer)
{
    const size_t blocksize = 256;
    const size_t numBlocks = (numEntries + blocksize - 1) / blocksize;

    const size_t size = numEntries * sizeof(uint8_t);

    for(int gpu=0;gpu<num_gpus;++gpu)
    {
        gpuErrorCheck(hipSetDevice(gpu));
        gpuErrorCheck(hipMalloc( (void **) &(gpuBuffer[gpu]), size));

        initialize_ker<<<numBlocks, blocksize>>>(numEntries,gpuBuffer[gpu]);
    }
    for(int numa=0;numa<num_numa;++numa)
    {
        hostBuffer[numa] =static_cast<uint8_t *>(numa_alloc_onnode(size,numa));
        gpuErrorCheck(hipHostRegister(hostBuffer[numa], size,\
				hipHostRegisterMapped));
        hipMemcpy(hostBuffer[numa],gpuBuffer[0],numEntries*sizeof(uint8_t),\
			hipMemcpyDefault);
        hipDeviceSynchronize();
    }
}

//Free memory and cleanup.
inline void cleanup(const int num_numa, const int num_gpus,\
		const size_t numEntries, uint8_t **gpuBuffer,\
		uint8_t **hostBuffer)
{
    const size_t size = numEntries * sizeof(uint8_t);


    for(int gpu=0;gpu<num_gpus;++gpu)
    {
        gpuErrorCheck(hipFree(gpuBuffer[gpu]));
    }
    for(int numa=0;numa<num_numa;++numa)
    {
        numa_free(hostBuffer[numa], size);
    }

    delete[] hostBuffer;
    delete[] gpuBuffer;
}

//Run a loop over cuda memcpy calls, measuring the bandwidth observed. 
inline void copy_loop(const int num_gpus,const int num_numa,const int warmup,\
		const int loops, const size_t numEntries, const bool time,\
		uint8_t ** gpuBuffer, uint8_t **hostBuffer, const int htod)
{
    const int size = numEntries * sizeof(uint8_t);
    for(int gpu=0;gpu<num_gpus;++gpu)
    {
        gpuErrorCheck(hipSetDevice(gpu));
        for(int numa=0;numa<num_numa;++numa)
        {
    	    hipEvent_t start,stop;
    	    hipEventCreate(&start);
    	    hipEventCreate(&stop);
            double totaltime = 0.0;
    	    float localtime = 0.0;

	    //run memcpy commands in a loop. the first loops will not be
	    //counted in any bandwidth calculation, they are just warmups.
    	    for(int iloop=0;iloop<warmup+loops;++iloop)
    	    {
	        hipError_t copy_stat;
    	        hipEventRecord(start,0);

	        //time host to device or device to host copies.
    		if(htod){copy_stat=hipMemcpyAsync(gpuBuffer[gpu],\
				hostBuffer[numa],size,hipMemcpyDefault); }
    		else{copy_stat=hipMemcpyAsync(hostBuffer[numa],\
				gpuBuffer[gpu],size,hipMemcpyDefault);}

    	        hipEventRecord(stop,0);
    	        hipEventSynchronize(stop);
		gpuErrorCheck(copy_stat);
    
    	        //touch the host and device buffers just so no compilers try to
		//optimize any of the copies away. This is probably 
		//unnecessary.
		dummy_kernel<<<1, 64>>>(gpuBuffer[gpu]);
    		gpuErrorCheck(hipMemcpyAsync(hostBuffer[numa],\
				gpuBuffer[gpu],sizeof(uint8_t),hipMemcpyDefault));
    
    	        hipEventElapsedTime(&localtime, start, stop);
    	        if(iloop>=warmup){totaltime += localtime;}
    	    }
    	    if(time){
	    //compute bandwidth in GBps
            double m1 = double(loops)/sizeof(uint8_t);
	    double m2 = double(numEntries)/1.0e6;
    	    printf("gpu%d_numa%d %f GBps \n",gpu,numa,m1*m2/totaltime);
    	    }
    	    hipDeviceSynchronize();
        }
    }
}


int main(int argc, char** argv)
{
    int ishtod=-1;
    size_t numEs = 0;
    if(argc > 1)
    {
	for(int j=1;j<argc;++j)
	{
	    //Is the test htod or dtoh, and what is the size of the test.
            int cmphtod = strcmp(argv[j],"--htod");
            int cmpdtoh = strcmp(argv[j],"--dtoh");
            if(cmphtod == 0){ishtod=1;}
            if(cmpdtoh == 0){ishtod=0;}

	    int cmpsize = strcmp(argv[j],"--size");
	    if(cmpsize == 0)
	    {
                j+=1;
		if(argc-1<j)
		{
	            printf("Missing an input to --size <number> \n");
		    exit(1);
		}
		char * ptr;
		long a = strtol(argv[j],&ptr,10);

	        if ((ptr == argv[j]) || (*ptr != '\0'))
		{
		    printf("Size input was not read correctly \n");
		    exit(1);
		}
		numEs = size_t(a);
	    }

	}
    }
    if((ishtod!=0) && (ishtod!=1))
    {
        printf("Unrecognized option. Please specify --htod or --dtoh.\n");
        return 1;
    }
    if((numEs==0))
    {
        printf("Must specify a size with --size <number>\n");
        return 1;
    }
		    
    // Get number of NUMA nodes
    if (numa_available()) 
    {
        fprintf(stderr, "main::numa_available error\n");
	printf("numa_available error failed");
        return -1;
    }
    int num_numa = numa_num_configured_nodes();
    int num_gpus = 0;
    
    //Get the number of visible GPUs, each will be tested.
    gpuErrorCheck(hipGetDeviceCount(&num_gpus));

    const size_t numEntries=numEs;

    //Allocate buffers to store the host and device buffers.
    uint8_t** hostBuffer = new uint8_t *[num_numa];
    uint8_t** gpuBuffer = new uint8_t *[num_gpus];

    //Initialize the host and device buffers.
    initialize(num_numa,num_gpus,numEntries,gpuBuffer,hostBuffer);

    //Set the number of warmup loops and loops that will be included in the 
    //bandwidth calculations.
    const int warmup=10;
    const int loops=20;

    //Run a set of warmup loops.
    copy_loop(num_gpus, num_numa, warmup, 0, numEntries, 0, gpuBuffer,\
		    hostBuffer, ishtod);
    //Run a set of loops used to calculate the bandwidth.
    copy_loop(num_gpus, num_numa, warmup, loops, numEntries, 1, gpuBuffer,\
		    hostBuffer, ishtod);
   
    //Deallocate memory and cleanup
    cleanup(num_numa,num_gpus,numEntries,gpuBuffer,hostBuffer);

    return 0;
}
